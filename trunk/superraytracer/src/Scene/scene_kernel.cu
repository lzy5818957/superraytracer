#include "scene_kernel.cuh"
#include <cstdio>
#include <cstdlib>

extern "C" RayTracing::HitInfo_t* hitInfoDTH(const RayTracing::HitInfo_t *devHitInfos, const int w, const int h)
{
	hipError_t cudaStatus;

	RayTracing::HitInfo_t* hostHitInfos;


	hostHitInfos = (RayTracing::HitInfo_t*)malloc(w * h * sizeof(RayTracing::HitInfo_t));

	cudaStatus = hipMemcpy(hostHitInfos, devHitInfos, w * h * sizeof(RayTracing::HitInfo_t), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	return hostHitInfos;

Error:

	printf("CUDA ERROR OCCURED\n");

	return NULL;
}

extern "C" RayTracing::Ray_t* rayHTD(const RayTracing::Ray_t *hostRays, const int w, const int h)
{
	hipError_t cudaStatus;

	RayTracing::Ray_t* devRays = 0;


	cudaStatus = hipMalloc (( void **)& devRays , 6 * w * h * sizeof ( float ));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(devRays, hostRays, 6 * w * h * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	return devRays;

Error:

	printf("CUDA ERROR OCCURED\n");

	return NULL;
}