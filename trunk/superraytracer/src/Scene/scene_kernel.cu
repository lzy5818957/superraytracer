#include "hip/hip_runtime.h"
#include <cutil_math.h>

#include <cstdio>
#include <cstdlib>
#include <cfloat>

#include "scene_kernel.cuh"

#define BLOCK_SIZE 8

__device__ void Mat4x4_Mul_Vec4_Scene(float *A, float *B, float *C)
{
	C[0] = A[0]*B[0]+A[4]*B[1]+A[8]*B[2]+A[12]*B[3]; 
	C[1] = A[1]*B[0]+A[5]*B[1]+A[9]*B[2]+A[13]*B[3];
	C[2] = A[2]*B[0]+A[6]*B[1]+A[10]*B[2]+A[14]*B[3];
	C[3] = A[3]*B[0]+A[7]*B[1]+A[11]*B[2]+A[15]*B[3];
}


__global__ void lightPropHTDKernel(const float* lightPos, const float* lightRad, float* lightProp)
{

	lightProp[0] = lightPos[0];
	lightProp[1] = lightPos[1];
	lightProp[2] = lightPos[2];

	lightProp[3] = lightRad[0];
	lightProp[4] = lightRad[1];
	lightProp[5] = lightRad[2];
}

__global__ void findClosestHitsKernel(RayTracing::HitInfo_t** hitInfos_array, const int w, const int h, RayTracing::HitInfo_t* closestHits, const int m_nObjects)
{
	int c = (blockIdx.x * blockDim.x) + threadIdx.x;
	int r = (blockIdx.y * blockDim.y) + threadIdx.y;
	int arrayPos1 = c + w * r;

	RayTracing::HitInfo_t closestHit = hitInfos_array[0][arrayPos1];

	for(int i = 1 ; i < m_nObjects; i++ )
	{
		if(hitInfos_array[i][arrayPos1].hitDist < closestHit.hitDist)
		{
			closestHit = hitInfos_array[i][arrayPos1];
		}

	}
	closestHits[arrayPos1] = closestHit;

}

__global__ void mergeShadowKernel(bool** isInShadow_array, const int w, const int h, bool* shadow, const int m_nObjects)
{
	int c = (blockIdx.x * blockDim.x) + threadIdx.x;
	int r = (blockIdx.y * blockDim.y) + threadIdx.y;
	int arrayPos1 = c + w * r;

	for(int i = 1 ; i < m_nObjects; i++ )
	{
		if(isInShadow_array[i][arrayPos1])
		{
			shadow[arrayPos1] = true;
			return;
		}

	}

}

__global__ void shadeRaysShadowLightKernel(
	const bool *isInShadow,
	const int w, const int h,
	float3* shades)
{
	int c = (blockIdx.x * blockDim.x) + threadIdx.x;
	int r = (blockIdx.y * blockDim.y) + threadIdx.y;
	int arrayPos1 = c + w * r;

	if(isInShadow[arrayPos1])
	{

		shades[arrayPos1] = make_float3(0.0f,0.0f,0.0f);
	}

	
}

__global__ void genShadowRaysKernel(
	const RayTracing::Ray_t *rays,
	const RayTracing::HitInfo_t *hitinfos,
	const float* lightProp,
	const int w, const int h,
	float *shadowRays)
{
	int c = (blockIdx.x * blockDim.x) + threadIdx.x;
	int r = (blockIdx.y * blockDim.y) + threadIdx.y;
	int arrayPos1 = c + w * r;
	int arrayPos6 = 6*(c + w * r);

	
	RayTracing::HitInfo_t hitInfo = hitinfos[arrayPos1];
	float3 rayOri = make_float3(rays[arrayPos1].o.x, rays[arrayPos1].o.y, rays[arrayPos1].o.z);
	float3 rayDir = make_float3(rays[arrayPos1].d.x, rays[arrayPos1].d.y, rays[arrayPos1].d.z);
	float3 shadePoint = rayOri + (hitInfo.hitDist * rayDir);
	float3 m_lightPos3 = make_float3(lightProp[0],lightProp[1],lightProp[2] );
	float3 shadowRayDir = normalize(m_lightPos3 - shadePoint);
	
	
	shadowRays[arrayPos6] = shadePoint.x;
	shadowRays[arrayPos6 + 1] = shadePoint.y;
	shadowRays[arrayPos6 + 2] = shadePoint.z;

	shadowRays[arrayPos6 + 3] = shadowRayDir.x;
	shadowRays[arrayPos6 + 4] = shadowRayDir.y;
	shadowRays[arrayPos6 + 5] = shadowRayDir.z;
	
}

__device__ float3 shadeLambPhone(	
	float3 lightRad, // Light radiance
	float3 lightDir, // Direction from point to light
	float3 e, // View direction
	float3 p, // Point being shaded (world-space)
	float3 n,
	float3 surfRefl) // Normal of p (world-space)
{
	float diff = dot(lightDir, n);
	if (diff <= 0.0)
	{
		return make_float3(0.0f,0.0f,0.0f);
	}

	float3 lamb = diff * (lightRad * surfRefl);

	return lamb;

}

__global__ void shadeRaysDirectLightKernel(
	const RayTracing::Ray_t *rays,
	const RayTracing::HitInfo_t *hitinfos,
	const RayTracing::Object_Kernel_t* objects,
	const float* lightProp,
	const int remainingRecursionDepth,
	const int w, const int h,
	float3* shades)
{
	int c = (blockIdx.x * blockDim.x) + threadIdx.x;
	int r = (blockIdx.y * blockDim.y) + threadIdx.y;
	int arrayPos1 = c + w * r;

	if(c == 350 && r == 340)
	{
		int arrayPos1 = c + w * r;
	}

	RayTracing::HitInfo_t hitInfo = hitinfos[arrayPos1];

	float3 color;
	if(hitInfo.hitDist == FLT_MAX)
	{
		color = make_float3(0.0f,0.0f,0.0f);
	}
	else
	{
		RayTracing::Object_Kernel_t object = objects[(int)hitInfo.objHit];
		RayTracing::GeometryType_Kernel geoType = object.m_geometry_type;

		RayTracing::Material_Kernel_t mat = object.m_material;

		RayTracing::ShaderType_Kernel shadeType = mat.m_shadeType;
		bool hasSpecular = mat.m_hasSpecular;

		//shared shader data
		float3 shadePoint = (*(float3*)&rays[arrayPos1].o) + hitInfo.hitDist * (*(float3*)&rays[arrayPos1].d);
		float3 m_lightPos3 = make_float3(lightProp[0],lightProp[1],lightProp[2] );
		float3 lightRad = make_float3(lightProp[3],lightProp[4],lightProp[5] );
		float3 lightDir = normalize(m_lightPos3-shadePoint); 
		float3 viewDir = (*(float3*)&rays[arrayPos1].d);
		//shader data;
		float3 normal;

		//ShaderV
		switch(object.m_geometry_type)
		{
		case RayTracing::GeometryType_Kernel::PLANE:
			normal = make_float3(0.0f,1.0f,0.0f);
			break;
		case RayTracing::GeometryType_Kernel::SPHERE:
			float3 shadePointObj = make_float3(hitInfo.sphere.shadePoint_x, hitInfo.sphere.shadePoint_y, hitInfo.sphere.shadePoint_z);
			normal = (1/hitInfo.hitDist) * shadePointObj;
			break;
		case RayTracing::GeometryType_Kernel::OCTAHEDRON:
			break;
		default:
			break;
		}

		float4 normalWorld;
		Mat4x4_Mul_Vec4_Scene(object.m_objectToWorld_Normals, (float*)&make_float4(normal,1.0f), (float*)&normalWorld);
		normal = normalize( make_float3( normalWorld ) );

		if(hasSpecular)
		{
			//get surface normal 
			color = make_float3(1.0f,1.0f,1.0f);
		}
		else
		{
			//shade use lamb phone
			
			color = shadeLambPhone(lightRad,lightDir,viewDir,shadePoint,normal, *((float3*)&mat.m_surfRefl));
			
		}


	}

	shades[arrayPos1] = color;
}


extern "C" float* rgbDTH(const float *devImg, const int w, const int h)
{
	hipError_t cudaStatus;

	float* hostImg;

	hostImg = (float*)malloc(w * h * sizeof(gml::vec3_t));

	cudaStatus = hipMemcpy(hostImg, devImg, w * h * sizeof(float3), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	hipFree((void*)devImg);
	return hostImg;

Error:

	printf("CUDA ERROR OCCURED\n");

	return NULL;
}

extern "C" bool* mergeShadowWithCuda(RayTracing::Ray_t *shadowRays, const bool** isInShadow_array, const int w, const int h, const int m_nObjects)
{	

	bool* shadow = 0;
	bool** devIsInShadow_array = 0;
	hipError_t cudaStatus;

	cudaStatus = hipMalloc (( void **)& devIsInShadow_array , m_nObjects * sizeof (bool* ));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(devIsInShadow_array, isInShadow_array, m_nObjects * sizeof(bool*), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMalloc (( void **)& shadow , w * h * sizeof ( bool ));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMemset (shadow ,0x00, w * h * sizeof ( bool ));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d before launching findClosestHits!\n", cudaStatus);
		goto Error;
	}

	dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);  // 64 threads 

	dim3 numBlocks(w/threadsPerBlock.x,  /* for instance 512/8 = 64*/ 
		h/threadsPerBlock.y);  

	mergeShadowKernel <<<numBlocks, threadsPerBlock>>>( devIsInShadow_array, w, h, shadow, m_nObjects );


	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching findClosestHits!\n", cudaStatus);
		goto Error;
	}

	for(int i = 0 ; i < m_nObjects; i++)
	{
		hipFree((void*)(isInShadow_array[i]));
		isInShadow_array[i] = 0;
	}

	hipFree(devIsInShadow_array);
	hipFree((void*)shadowRays);

	return shadow;
	

Error:

	hipFree(devIsInShadow_array);
	printf("CUDA ERROR OCCURED\n");

	return NULL;
}

extern "C" RayTracing::HitInfo_t* findClosestHitsWithCuda(const RayTracing::HitInfo_t** hitInfos_array, const int w, const int h, const int m_nObjects)
{	

	RayTracing::HitInfo_t* closestHits = 0;
	RayTracing::HitInfo_t** devHitInfos_array = 0;
	hipError_t cudaStatus;

	cudaStatus = hipMalloc (( void **)& devHitInfos_array , m_nObjects * sizeof ( RayTracing::HitInfo_t* ));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(devHitInfos_array, hitInfos_array, m_nObjects * sizeof(RayTracing::HitInfo_t*), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMalloc (( void **)& closestHits , w * h * sizeof ( RayTracing::HitInfo_t ));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d before launching findClosestHits!\n", cudaStatus);
		goto Error;
	}

	dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);  // 64 threads 

	dim3 numBlocks(w/threadsPerBlock.x,  /* for instance 512/8 = 64*/ 
		h/threadsPerBlock.y);  

	findClosestHitsKernel <<<numBlocks, threadsPerBlock>>>( devHitInfos_array, w, h, closestHits, m_nObjects );


	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching findClosestHits!\n", cudaStatus);
		goto Error;
	}

	for(int i = 0 ; i < m_nObjects; i++)
	{
		hipFree((void*)(hitInfos_array[i]));
		hitInfos_array[i] = 0;
	}

	hipFree(devHitInfos_array);
	return closestHits;


Error:

	hipFree(devHitInfos_array);
	printf("CUDA ERROR OCCURED\n");

	return NULL;
}



extern "C" float* shadeRaysDirectLightWithCuda(
	const RayTracing::Ray_t *rays,
	const RayTracing::HitInfo_t *hitinfos,
	const RayTracing::Object_Kernel_t* objects,
	const float* lightProp,
	const int remainingRecursionDepth,
	const int w, const int h)
{
	hipError_t cudaStatus;
	float3* devShades = 0;

	cudaStatus = hipMalloc (( void **)& devShades , w * h * sizeof ( float3 ));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d before launching shadeRaysKernel!\n", cudaStatus);
		goto Error;
	}


	dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);  // 64 threads 

	dim3 numBlocks(w/threadsPerBlock.x,  /* for instance 512/8 = 64*/ 
		h/threadsPerBlock.y);

	shadeRaysDirectLightKernel <<<numBlocks, threadsPerBlock>>>( rays,hitinfos, objects, lightProp, remainingRecursionDepth,w,h,devShades);


	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching shadeRaysKernel!\n", cudaStatus);
		goto Error;
	}

	return (float*)devShades;

Error:

	printf("CUDA ERROR OCCURED\n");
	return NULL;
}

extern "C" float* shadeRaysShadowLightWithCuda
	(
	const bool *isInShadow,
	const int w, const int h,
	float* colors
	)
{
	hipError_t cudaStatus;

	
	dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);  // 64 threads 

	dim3 numBlocks(w/threadsPerBlock.x,  /* for instance 512/8 = 64*/ 
		h/threadsPerBlock.y);

	shadeRaysShadowLightKernel <<<numBlocks, threadsPerBlock>>>( isInShadow,w,h,(float3*)colors);


	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching shadeRaysKernel!\n", cudaStatus);
		goto Error;
	}
	hipFree((void*)isInShadow);
	isInShadow = 0;

	return colors;
Error:

	printf("CUDA ERROR OCCURED\n");
	return NULL;

}

extern "C" RayTracing::Ray_t* genShadowRaysWithCuda
	(
	const RayTracing::Ray_t *rays,
	const RayTracing::HitInfo_t *hitinfos,
	const float *lightProp,
	const int w, const int h
	)
{
	hipError_t cudaStatus;

	float* shadowRays = 0;

	cudaStatus = hipMalloc (( void **)& shadowRays , 2 * w * h * sizeof (float3));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	
	dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);  // 64 threads 

	dim3 numBlocks(w/threadsPerBlock.x,  /* for instance 512/8 = 64*/ 
		h/threadsPerBlock.y);

	genShadowRaysKernel <<<numBlocks, threadsPerBlock>>>(rays, hitinfos, lightProp, w, h, shadowRays);


	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching genShadowRaysKernel!\n", cudaStatus);
		goto Error;
	}
	return ( RayTracing::Ray_t*)shadowRays;

Error:

	printf("CUDA ERROR OCCURED\n");
	return NULL;

}


extern "C" RayTracing::Ray_t* raysDTH(const RayTracing::Ray_t *rays, const int w, const int h)
{
	hipError_t cudaStatus;

	RayTracing::Ray_t* hostRays = 0;
	hostRays = (RayTracing::Ray_t*)malloc( w * h * sizeof( RayTracing::Ray_t));


	cudaStatus = hipMemcpy(hostRays, rays, w * h * sizeof( RayTracing::Ray_t), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	/*
	for(int i = 0; i < w * h; i++)
	{
		printf("Num = %d  %f &f %f %f %f %f\n",i, hostRays[i].o.x, hostRays[i].o.y, hostRays[i].o.z, hostRays[i].d.x, hostRays[i].d.y, hostRays[i].d.z);

	}
	*/
	return hostRays;

Error:
	printf("CUDA ERROR OCCURED\n");
	return NULL;
}

extern "C" RayTracing::Object_Kernel_t* objHTD(const RayTracing::Object_Kernel_t *hostObj, const int m_nObjects)
{
	hipError_t cudaStatus;

	RayTracing::Object_Kernel_t* devObjs = 0;
	cudaStatus = hipMalloc (( void **)& devObjs , m_nObjects * sizeof ( RayTracing::Object_Kernel_t ));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(devObjs, hostObj, m_nObjects * sizeof(RayTracing::Object_Kernel_t), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	return devObjs;

Error:
	printf("CUDA ERROR OCCURED\n");
	return NULL;
}

extern "C" float* lightPropHTD(	const float* lightPos,const float* lightRad, const int w, const int h)
{
	hipError_t cudaStatus;
	float* devLightPos = 0;
	float* devLightRad = 0;
	float* devLightProp = 0;

	cudaStatus = hipMalloc (( void **)& devLightProp ,2 * sizeof ( float3 ));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc (( void **)& devLightPos ,sizeof ( float3 ));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	
	cudaStatus = hipMemcpy(devLightPos, lightPos, sizeof(float3), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMalloc (( void **)& devLightRad ,sizeof ( float3 ));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(devLightRad, lightRad, sizeof(float3), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	lightPropHTDKernel<<<1,1>>>(devLightPos,devLightRad,devLightProp);

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching shadeRaysKernel!\n", cudaStatus);
		goto Error;
	}

	hipFree(devLightPos);
	devLightPos = 0;

	hipFree(devLightRad);
	devLightRad = 0;

	return (float*)devLightProp;

Error:
	printf("CUDA ERROR OCCURED\n");
	return NULL;
}

extern "C" void cleanUp(void** handles, int nHandles)
{
	for(int i = 0 ; i < nHandles; i++)
	{
		hipFree(handles[i]);
		handles[i] = 0;
	}

}