#include "hip/hip_runtime.h"
#include "scene_kernel.cuh"
#include <cstdio>
#include <cstdlib>

#define BLOCK_SIZE 8

__global__ void findClosestHitsKernel(RayTracing::HitInfo_t** hitInfos_array, const int w, const int h, RayTracing::HitInfo_t* closestHits, const int m_nObjects)
{
	int c = (blockIdx.x * blockDim.x) + threadIdx.x;
	int r = (blockIdx.y * blockDim.y) + threadIdx.y;
	int arrayPos1 = c + w * r;
	
	RayTracing::HitInfo_t closestHit = hitInfos_array[0][arrayPos1];

	for(int i = 1 ; i < m_nObjects; i++ )
	{
		if(hitInfos_array[i][arrayPos1].hitDist < closestHit.hitDist)
		{
			closestHit = hitInfos_array[i][arrayPos1];
		}
		
	}
	closestHits[arrayPos1] = closestHit;

}

__global__ void shadeRaysKernel(const RayTracing::Ray_t *rays, RayTracing::HitInfo_t *hitinfos, const int remainingRecursionDepth, const int w, const int h, float3* shades)
{
	int c = (blockIdx.x * blockDim.x) + threadIdx.x;
	int r = (blockIdx.y * blockDim.y) + threadIdx.y;
	int arrayPos1 = c + w * r;
	shades[arrayPos1] = make_float3(1.0f,0.0f,0.0f);
}


extern "C" float* rgbDTH(const float *devImg, const int w, const int h)
{
	hipError_t cudaStatus;

	float* hostImg;

	hostImg = (float*)malloc(w * h * sizeof(gml::vec3_t));
	
	cudaStatus = hipMemcpy(hostImg, devImg, w * h * sizeof(float3), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	
	hipFree((void*)devImg);
	return hostImg;

Error:

	printf("CUDA ERROR OCCURED\n");

	return NULL;
}


extern "C" RayTracing::HitInfo_t* findClosestHitsWithCuda(const RayTracing::HitInfo_t** hitInfos_array, const int w, const int h, const int m_nObjects)
{	

	RayTracing::HitInfo_t* closestHits = 0;
	RayTracing::HitInfo_t** devHitInfos_array = 0;
	hipError_t cudaStatus;

	cudaStatus = hipMalloc (( void **)& devHitInfos_array , m_nObjects * sizeof ( RayTracing::HitInfo_t* ));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(devHitInfos_array, hitInfos_array, m_nObjects * sizeof(RayTracing::HitInfo_t*), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMalloc (( void **)& closestHits , w * h * sizeof ( RayTracing::HitInfo_t ));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d before launching findClosestHits!\n", cudaStatus);
		goto Error;
	}

	dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);  // 64 threads 

	dim3 numBlocks(w/threadsPerBlock.x,  /* for instance 512/8 = 64*/ 
		h/threadsPerBlock.y);  
	
	findClosestHitsKernel <<<numBlocks, threadsPerBlock>>>( devHitInfos_array, w, h, closestHits, m_nObjects );


	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching findClosestHits!\n", cudaStatus);
		goto Error;
	}

	size_t free, total;

	hipMemGetInfo(&free,&total);
	printf("before:      avaliable mem = %lu\n", free);

	for(int i = 0 ; i < m_nObjects; i++)
	{

		hipFree((void*)(hitInfos_array[i]));
		hitInfos_array[i] = 0;


	}
	hipMemGetInfo(&free,&total);
	printf("after free:  avaliable mem = %lu\n", free);
	printf("--------------------------------\n");

	hipFree(devHitInfos_array);
	return closestHits;
	

Error:

	hipFree(devHitInfos_array);
	printf("CUDA ERROR OCCURED\n");

	return NULL;
}

extern "C" float* shadeRaysWithCuda(const RayTracing::Ray_t *rays, RayTracing::HitInfo_t *hitinfos, const int remainingRecursionDepth, const int w, const int h)
{
	hipError_t cudaStatus;
	float3* devShades = 0;
	cudaStatus = hipMalloc (( void **)& devShades , w * h * sizeof ( float3 ));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);  // 64 threads 

	dim3 numBlocks(w/threadsPerBlock.x,  /* for instance 512/8 = 64*/ 
		h/threadsPerBlock.y);
	
	shadeRaysKernel <<<numBlocks, threadsPerBlock>>>( rays,hitinfos,remainingRecursionDepth,w,h,devShades );

	hipFree((void*)rays);
	rays = 0;

	hipFree((void*)hitinfos);
	hitinfos = 0;
	

	return (float*)devShades;
	
Error:

	printf("CUDA ERROR OCCURED\n");
	return NULL;
}