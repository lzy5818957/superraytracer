#include "plane_kernel.cuh"
#include <cstdio>

extern "C" hipError_t raysIntersectsWithCudaPlane(float *devRays, const float t0, const float t1, const int w, const int h, RayTracing::HitInfo_t *hostHitInfos)
{
	RayTracing::HitInfo_t *devHitInfos = 0;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}


	cudaStatus = hipMalloc (( void **)& devHitInfos , w * h * sizeof ( RayTracing::HitInfo_t ));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}



Error:
	//hipFree(devRayDirs);
	return cudaStatus;
}