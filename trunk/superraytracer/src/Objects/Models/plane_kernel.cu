#include "plane_kernel.cuh"
#include <cstdio>

#define BLOCK_SIZE 8

extern "C" hipError_t raysIntersectsWithCudaPlane(float *devRays, const float t0, const float t1, const int w, const int h, RayTracing::HitInfo_t *hostHitInfos)
{
	RayTracing::HitInfo_t *devHitInfos = 0;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}


	cudaStatus = hipMalloc (( void **)& devHitInfos , w * h * sizeof ( RayTracing::HitInfo_t ));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);  // 64 threads 

	dim3 numBlocks(w/threadsPerBlock.x,  /* for instance 512/8 = 64*/ 
		h/threadsPerBlock.y);  


Error:
	//hipFree(devRayDirs);
	return cudaStatus;
}