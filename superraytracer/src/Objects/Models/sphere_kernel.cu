#include "hip/hip_runtime.h"
#include "sphere_kernel.cuh"
#include <cstdio>
#include <cfloat>
#include <cutil_math.h>

#define BLOCKSIZE 8

__global__ void raysIntersectsSphereKernel(float *devRays, const float t0, const float t1,const int w, const int h,RayTracing::HitInfo_t *hitInfos, int objHitIndex)
{
	float A,B,C;
	int c = (blockIdx.x * blockDim.x) + threadIdx.x;
	int r = (blockIdx.y * blockDim.y) + threadIdx.y;
	int arrayPos1 = c + w * r;
	int arrayPos6 = 6 * (c + w * r);

	float3 ray_o;
	ray_o.x = devRays[arrayPos6];
	ray_o.y = devRays[arrayPos6 +1];
	ray_o.z = devRays[arrayPos6 +2];

	float3 ray_d;
	ray_d.x = devRays[arrayPos6 +3];
	ray_d.y = devRays[arrayPos6 +4];
	ray_d.z = devRays[arrayPos6 +5];

	A = dot(ray_d,ray_d);
	B = dot(ray_d,ray_o);
	C = dot(ray_o,ray_o) - 1.0f;

	float det = B*B - A*C;

	if(det < 0.0)
	{
		hitInfos[arrayPos1].hitDist = FLT_MAX;
	}else
	{
		
		float t =  (-B - sqrt(B * B - A * C)) / A;
	    if(t > t1 || t < t0 )
	    {

	       hitInfos[arrayPos1].hitDist = FLT_MAX;
	    }
	    else
	    {
 	       hitInfos[arrayPos1].hitDist = t;
		   float3 shadePoint = ray_o + (t * ray_d);
		   hitInfos[arrayPos1].sphere.shadePoint_x = shadePoint.x;
		   hitInfos[arrayPos1].sphere.shadePoint_y = shadePoint.y;
		   hitInfos[arrayPos1].sphere.shadePoint_z = shadePoint.z;
		   hitInfos[arrayPos1].objHit = (Object::Object*)objHitIndex;

	    }
		


	}
}


__global__ void hitPropertiesSphereKernel(const RayTracing::HitInfo_t *hitinfos,  const int w, const int h,float *normTex)
{
	int c = (blockIdx.x * blockDim.x) + threadIdx.x;
	int r = (blockIdx.y * blockDim.y) + threadIdx.y;
	int arrayPos1 = (c + w * r);
	int arrayPos5 = 5 * (c + w * r);

	float3 shadePoint;
	float3 normal;
	float2 texCoords;

	shadePoint.x = hitinfos[arrayPos1].sphere.shadePoint_x;
	shadePoint.y = hitinfos[arrayPos1].sphere.shadePoint_y;
	shadePoint.z = hitinfos[arrayPos1].sphere.shadePoint_z;

	normal = hitinfos[arrayPos1].hitDist * shadePoint;
	texCoords.x = (atan2 ( shadePoint.z, - shadePoint.x) / 3.14159265358979323846 +1 )/ 2.0f;
	texCoords.y = ( asin ( -shadePoint.y ) / 3.14159265358979323846 +1)/ 2;

	normTex [arrayPos5] = normal.x;
	normTex [arrayPos5 + 1] = normal.y;
	normTex [arrayPos5 + 2] = normal.z;
	normTex [arrayPos5 + 3] = texCoords.x;
	normTex [arrayPos5 + 4] = texCoords.y;
}

extern "C" RayTracing::HitInfo_t* raysIntersectsWithCudaSphere(float *devRays, const float t0, const float t1,const int w, const int h,  int objHitIndex)
{
	RayTracing::HitInfo_t *devHitInfos = 0;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	cudaStatus = hipMalloc (( void **)& devHitInfos , w * h * sizeof ( RayTracing::HitInfo_t ));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d before launching setup_rand_kernel!\n", cudaStatus);
		goto Error;
	}

	dim3 threadsPerBlock(BLOCKSIZE,BLOCKSIZE);
	dim3 numBlocks(w/threadsPerBlock.x,h/threadsPerBlock.y);

	raysIntersectsSphereKernel <<< numBlocks, threadsPerBlock>>> (devRays,t0,t1,w,h,devHitInfos, objHitIndex);

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d before launching setup_rand_kernel!\n", cudaStatus);
		goto Error;
	}

	hipFree(devRays);
	devRays = 0;

	return devHitInfos;
	

Error:
	hipFree(devHitInfos);
	devHitInfos = 0;
	return NULL;

}

extern "C" float* hitPropertiesWithCudaSphere(const RayTracing::HitInfo_t*hitinfos, const int w, const int h)
{
	float *devNormTex = 0;
	hipError_t cudaStatus;

	cudaStatus = hipMalloc((void **)& devNormTex , 5 * w * h * sizeof (float));
	if(cudaStatus != hipSuccess)
	{
		fprintf(stderr,"hipMalloc failed! ");
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d before launching setup_rand_kernel!\n", cudaStatus);
		goto Error;
	}

	dim3 threadsPerBlock(BLOCKSIZE,BLOCKSIZE);
	dim3 numBlocks(w/threadsPerBlock.x,h/threadsPerBlock.y);

	hitPropertiesSphereKernel<<<numBlocks,threadsPerBlock>>>(hitinfos,w,h,devNormTex);

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d before launching setup_rand_kernel!\n", cudaStatus);
		goto Error;
	}

	return devNormTex;
	
Error:
	hipFree(devNormTex);
	return NULL;


}