#include "hip/hip_runtime.h"
#include "plane_kernel.cuh"
#include <cstdio>

#define BLOCK_SIZE 8

__global__ void raysIntersectsPlaneKernel(float *devRays, const float t0, const float t1, const int w, const int h, RayTracing::HitInfo_t *hitInfos)
{


}

extern "C" hipError_t raysIntersectsWithCudaPlane(float *devRays, const float t0, const float t1, const int w, const int h, RayTracing::HitInfo_t *hostHitInfos)
{
	RayTracing::HitInfo_t *devHitInfos = 0;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	cudaStatus = hipMalloc (( void **)& devHitInfos , w * h * sizeof ( RayTracing::HitInfo_t ));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d before launching setup_rand_kernel!\n", cudaStatus);
		goto Error;
	}

	dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);  // 64 threads 

	dim3 numBlocks(w/threadsPerBlock.x,  /* for instance 512/8 = 64*/ 
		h/threadsPerBlock.y);  

	raysIntersectsPlaneKernel <<<numBlocks, threadsPerBlock>>>(devRays, t0, t1, w, h, devHitInfos);

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(hostHitInfos, devHitInfos, w * h * sizeof( RayTracing::HitInfo_t), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	hipFree(devHitInfos);
	devRays = 0;

Error:
	//hipFree(devRayDirs);
	return cudaStatus;
}