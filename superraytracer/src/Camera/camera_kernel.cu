#include "hip/hip_runtime.h"
#include "camera_kernel.cuh"
#include <cstdio>


__global__ void genViewRayKernel(RayTracing::Ray_t *rays)
{
    int c = (blockIdx.x * blockDim.x) + threadIdx.x;
	int r = (blockIdx.y * blockDim.y) + threadIdx.y;

	printf("%d, %d\n", c ,r);
}


extern "C" hipError_t genViewRayWithCuda(RayTracing::Ray_t *host_rays, const int w, const int h)
{

    RayTracing::Ray_t *dev_rays = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_rays, w * h * sizeof(RayTracing::Ray_t));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

	dim3 threadsPerBlock(8, 8);  // 64 threads 

	dim3 numBlocks(w/threadsPerBlock.x,  /* for instance 512/8 = 64*/ 
                h/threadsPerBlock.y);  


    // Launch a kernel on the GPU with one thread for each element.
    genViewRayKernel<<<numBlocks, threadsPerBlock>>>(dev_rays);

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(host_rays, dev_rays, w * h * sizeof(RayTracing::Ray_t), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_rays);
    
    return cudaStatus;

}
