#include "hip/hip_runtime.h"
#include "camera_kernel.cuh"
#include "hiprand/hiprand_kernel.h"
#include <cstdio>

__global__ void setup_rand_kernel ( hiprandState * state , int w, int h)
{
	int c = (blockIdx.x * blockDim.x) + threadIdx.x;
	int r = (blockIdx.y * blockDim.y) + threadIdx.y;
	/* Each thread gets same seed , a different sequence
	number , no offset */
	hiprand_init (1234 , 3 * c + w * r, 0, & state [  3 * c + w * r ]);
	hiprand_init (1234 , 1 + 3 * c + w * r, 0, & state [ 1 + 3 * c + w * r ]);
}
__global__ void generate_rand_kernel ( hiprandState *state ,	float *result, int w, int h )
{
	int c = (blockIdx.x * blockDim.x) + threadIdx.x;
	int r = (blockIdx.y * blockDim.y) + threadIdx.y;

	
	hiprandState localState = state [3 * c + w * r ];
	/* Store results */
	result [3 * c + w * r ] = hiprand_uniform (& localState );

	/* Copy state to local memory for efficiency */
	localState = state [1 + 3 * c + w * r ];
	result [ 1 + 3 * c + w * r ] = hiprand_uniform (& localState );


}

__global__ void genViewRayKernel(float *rayDirs, float* rand_result, int w, int h)
{
	int c = (blockIdx.x * blockDim.x) + threadIdx.x;
	int r = (blockIdx.y * blockDim.y) + threadIdx.y;


	float screenPositionInWorld4;


	const float x = c - 0.5 + rand_result[ 3 * c + w * r], y = r - 0.5 + rand_result[1 + 3 * c + r * w];

	float screenPos[4] = {x, y, 1, 1};

	/*
	screenPositionInWorld4 = gml::mul(m_windowToWorld, gml::vec4_t(x, y, 1, 1));
	gml::vec3_t screenPositionInWorld3 = gml::vec3_t(screenPositionInWorld4.x/screenPositionInWorld4.w,
		screenPositionInWorld4.y/screenPositionInWorld4.w,
		screenPositionInWorld4.z/screenPositionInWorld4.w);
*/
	//ray.d = gml::normalize(gml::sub(screenPositionInWorld3,ray.o));

	rayDirs[ 3 * c + w * r] = x;
	rayDirs[ 1 + 3 * c + r * w ] = y;
	rayDirs[ 2 + 3 * c + r * w ] = 0.0f;

}


extern "C" hipError_t genViewRayWithCuda(float *hostRayDirs, const int w, const int h)
{

	float *devRayDirs = 0;
	hiprandState * devStates;
	float *dev_rand_result;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&devRayDirs, w * h * 3 * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	/*
	hipMemset ( devRayDirs , 0, w * h * 3 * sizeof ( float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemset failed!");
		goto Error;
	}
	*/
	cudaStatus = hipMalloc (( void **)& dev_rand_result , 2 * w * h * sizeof ( float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	hipMemset ( dev_rand_result , 0, 2 * w * h * sizeof ( int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemset failed!");
		goto Error;
	}

	hipMalloc (( void **)& devStates , 2 * w * h * sizeof ( hiprandState ));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	dim3 threadsPerBlock(8, 8);  // 64 threads 

	dim3 numBlocks(w/threadsPerBlock.x,  /* for instance 512/8 = 64*/ 
		h/threadsPerBlock.y);  
	
	setup_rand_kernel <<<numBlocks, threadsPerBlock>>>( devStates, w, h );
	

	generate_rand_kernel <<<numBlocks, threadsPerBlock>>>( devStates , dev_rand_result, w, h );
	

	hipFree(devStates);
	// Launch a kernel on the GPU with one thread for each element.
	genViewRayKernel<<<numBlocks, threadsPerBlock>>>(devRayDirs, dev_rand_result, w, h);
	
	hipFree(dev_rand_result);

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(hostRayDirs, devRayDirs, 3 * w * h * sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(devRayDirs);
	return cudaStatus;

}
